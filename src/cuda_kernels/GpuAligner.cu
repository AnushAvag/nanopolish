#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "GpuAligner.h"
#include <vector>
#include "nanopolish_profile_hmm_r9.h"

#define MAX_STATES 1024

__device__ float logsumexpf(float x, float y){
    return fmax(x, y) + log1pf(expf(-fabsf(y-x)));
}

//TODO: Implement, inc pore model
__device__ float lp_match_r9(int rank,
                             float mean,
                             float * poreModelLevelLogStdv,
                             float * poreModelLevelStdv,
                             float * poreModelLevelMean){

    float log_inv_sqrt_2pi = log(0.3989422804014327); // no need to calculate this every time. better solutions available..

    // STEP 1: GET DRIFT-SCALED LEVEL:
    float level = mean; //TODO: Do actual drift scaling. this is a cheat
    // TODO: Apply scaling to these 3 model values as is done in the CPP implementation
    //these can just be pulled from the model
    float gaussian_mean = poreModelLevelMean[rank];
    float gaussian_stdv = poreModelLevelStdv[rank];
    float gaussian_log_level_stdv = poreModelLevelLogStdv[rank];
    // Step 3: calculate log-normal PDF
    float a = (level - gaussian_mean) / gaussian_stdv; // g is the gaussian parameters
    return log_inv_sqrt_2pi - gaussian_log_level_stdv + (-0.5f * a * a); // log_inv_sqrt_2pi is defined in a comment above
}

__global__ void getScores(float * eventData,
                          float * readEventsPerBase,
                          int * numRowsPerRead,
                          int * eventStarts,
                          int * eventStrides,
                          int * kmer_ranks,
                          int * kmer_ranks_rc,
                          int * eventOffsets, // Offset to use for getting an event IDX for a specific read (read obtained by block IDX)
                          float * poreModelLevelLogStdv,
                          float * poreModelLevelStdv,
                          float * poreModelLevelMean,
                          float * returnValues)
{
    // kmer probabilities will be stored here
    __shared__ float prevProbabilities[MAX_STATES];
    for (int i =0;i<MAX_STATES;i++){
        prevProbabilities[i] = -INFINITY;
    }

    //float log_inv_sqrt_2pi = log(0.3989422804014327);

    //Step 1: calculate transitions. For now we are going to use external params.
    int readIdx = blockIdx.x;
    float read_events_per_base = readEventsPerBase[readIdx];
    int numRows = numRowsPerRead[readIdx]; // Number of rows in this DP table.
    int e_start = eventStarts[readIdx]; // Event start for read
    int e_stride = eventStrides[readIdx];
    int e_offset = eventOffsets[readIdx]; // Within the event means etc, the offset needed for this block to get a specific event
    //int kmer_ranks = kmerRanks[readIdx.x]; // TODO: Use RC for RC reads

    int kmerIdx = threadIdx.x;

    float p_stay = 1 - (1 / read_events_per_base);

    //printf("Events per base: %f \n", read_events_per_base);
    float p_skip = 0.0025;
    float p_bad = 0.001;
    float p_bad_self = p_bad;
    float p_skip_self = 0.3;

    float p_mk = p_skip; // probability of not observing an event at all
    float p_mb = p_bad; // probabilty of observing a bad event
    float p_mm_self = p_stay; // probability of observing additional events from this k-mer
    float p_mm_next = 1.0f - p_mm_self - p_mk - p_mb; // normal movement from state to state

    // transitions from event split state in previous block
    float p_bb = p_bad_self;
    float p_bk, p_bm_next, p_bm_self;
    p_bk = p_bm_next = p_bm_self = (1.0f - p_bb) / 3;

    // transitions from kmer skip state in previous block
    float p_kk = p_skip_self;
    float p_km = 1.0f - p_kk;

    // We assign some transition probabilities. I believe this is correct and they don't vary by location in the sequence (why would they)
    float lp_mk = log(p_mk);
    float lp_mb = log(p_mb);
    float lp_mm_self = log(p_mm_self);
    float lp_mm_next = log(p_mm_next);
    float lp_bb = log(p_bb);
    float lp_bk = log(p_bk);
    float lp_bm_next = log(p_bm_next);
    float lp_bm_self = log(p_bm_self);
    float lp_kk = log(p_kk);
    float lp_km = log(p_km);


    // Start filling out the "DP table"
    // Each thread is going to work on an individual P-HMM Block
    // WRONG - need to use threadIdx & think carefully. we have one thread per block/kmer. each block has 3 states tho.
    //int kmerIdx = blockIdx.x;
    int curBlockIdx = kmerIdx + 1; // Accounts for fact that we are not working with start block.
    int prevBlockIdx = curBlockIdx -1;
    int prevBlockOffset = PSR9_NUM_STATES * prevBlockIdx;
    int curBlockOffset = PSR9_NUM_STATES * curBlockIdx;

    // the penalty is controlled by the transition probability
    float BAD_EVENT_PENALTY = 0.0f;

    for(int row=1; row<numRows;row++){
        // Emission probabilities
        int event_idx = e_start + (row - 1) * e_stride;
        uint32_t rank = kmer_ranks[kmerIdx]; // lexical rank of a kmer
        float event_mean = eventData[e_offset + row];
        float lp_emission_m = lp_match_r9(rank,
                                          event_mean,
                                          poreModelLevelLogStdv,
                                          poreModelLevelStdv,
                                          poreModelLevelMean);
        float lp_emission_b = BAD_EVENT_PENALTY;

        // Get all the scores for a match
        float HMT_FROM_SAME_M = lp_mm_self + prevProbabilities[curBlockOffset + PSR9_MATCH];
        float HMT_FROM_PREV_M = lp_mm_next + prevProbabilities[prevBlockOffset + PSR9_MATCH];
        float HMT_FROM_SAME_B = lp_bm_self + prevProbabilities[curBlockOffset + PSR9_BAD_EVENT];
        float HMT_FROM_PREV_B = lp_bm_next + prevProbabilities[prevBlockOffset + PSR9_BAD_EVENT];
        float HMT_FROM_PREV_K = lp_km + prevProbabilities[prevBlockOffset + PSR9_KMER_SKIP];

        // m_s is the probability of going from the start state
        // to this kmer. The start state is (currently) only
        // allowed to go to the first kmer. If ALLOW_PRE_CLIP
        // is defined, we allow all events before this one to be skipped,
        // with a penalty;
        // TODO: Implemnet the HMT_FROM_SOFT score. this appears needed but I don't yet understand it.

        // NOW calculate the score
        float sum = HMT_FROM_SAME_M;
        sum = logsumexpf(sum, HMT_FROM_PREV_M);
        sum = logsumexpf(sum, HMT_FROM_SAME_B);
        sum = logsumexpf(sum, HMT_FROM_PREV_B);
        sum = logsumexpf(sum, HMT_FROM_PREV_K);
        sum += lp_emission_m;

        __syncthreads();
        prevProbabilities[curBlockIdx + PSR9_MATCH] = sum;
        __syncthreads();
    }


    returnValues[blockIdx.x] = 0.356;
    __syncthreads();
}


GpuAligner::GpuAligner()
{
    y = 20;
    asize = y*sizeof(int);
    for (int i=0; i<y; i++)
        n[i] = i;
}

double scoreKernel(std::vector<HMMInputSequence> sequences,
                   std::vector<HMMInputData> event_sequences,
                   uint32_t alignment_flags){

    // Extract the pore model.
    //Let's assume that every event sequence has the same pore model
    //event_sequences[0].pore_model.

    int num_reads = event_sequences.size();
    // These asserts are here during the development phase
    assert(!sequences.empty());
    assert(std::string(sequences[0].get_alphabet()->get_name()) == "nucleotide");
    for (auto e: event_sequences) {
        assert(std::string(e.pore_model->pmalphabet->get_name()) == "nucleotide");
        assert(e.read->pore_type == PT_R9);
        assert( (e.rc && e.event_stride == -1) || (!e.rc && e.event_stride == 1));
    }

    size_t num_models = sequences.size();
    double num_model_penalty = log(num_models);

    assert(num_models == 1); //this is temporary

    auto sequence = sequences[0]; // temporary. We are only going to score one sequence against a set of events for now.

    const uint32_t k = event_sequences[0].pore_model->k; //k is the kmerity
    uint32_t n_kmers = sequence.length() - k + 1; //number of kmers in the sequence

    uint32_t n_states = PSR9_NUM_STATES * (n_kmers + 2); // + 2 for explicit terminal states

    std::vector<uint32_t> n_rows; //number of rows in the DP table (n_events + 1)
    std::vector<uint32_t> e_starts; //event starts
    std::vector<uint32_t> event_strides;

    for(auto e: event_sequences){
        uint32_t e_start = e.event_start_idx;
        e_starts.push_back(e_start);

        uint32_t e_stride = e.event_stride;
        event_strides.push_back(e_stride);

        uint32_t e_end = e.event_stop_idx;
        uint32_t n_events = 0;
        if(e_end > e_start)
            n_events = e_end - e_start + 1;
        else
            n_events = e_start - e_end + 1;

        n_rows.push_back(n_events + 1);
    }

    std::vector<uint32_t> kmer_ranks(n_kmers);
    std::vector<uint32_t> kmer_ranks_rc(n_kmers);
    for(size_t ki = 0; ki < n_kmers; ++ki) {
        kmer_ranks[ki] = sequences[0].get_kmer_rank(ki, k, false);
        kmer_ranks_rc[ki] = sequences[0].get_kmer_rank(ki, k, true);
    }

    // Prepare raw data and send it over to the score calculator kernel

    // Buffer 1: Raw event data and associated starts and stops

    size_t numEventsTotal = 0;
    //1. Count the total number of events across all reads
    std::vector<int> eventLengths;
    std::vector<float> eventsPerBase;
    for (auto e: event_sequences){
        size_t numEvents = e.read->events->size();
        float readEventsPerBase = e.read->events_per_base[e.strand];

        eventLengths.push_back(numEvents);
        eventsPerBase.push_back(readEventsPerBase);

        numEventsTotal += numEvents;
    }

    //Allocate a host buffer to store the event means
    float * eventMeans;
    size_t eventMeansSize = numEventsTotal * sizeof(float);
    hipHostAlloc(&eventMeans, eventMeansSize , hipHostMallocDefault);

    std::vector<int> eventOffsets;
    size_t offset = 0;
    for (auto ev: event_sequences){
        eventOffsets.push_back(offset);
        size_t num_events = ev.read->events->size();
        for (int i=0;i<num_events;i++) {
            eventMeans[offset + i] = ev.read->events[0][i].mean; //taking the first element. Not sure what the second one is..
        }
        offset += num_events;
    }

    int num_states = event_sequences[0].pore_model->states.size();

    std::vector<float> pore_model_level_log_stdv(num_states);
    std::vector<float> pore_model_level_mean(num_states);
    std::vector<float> pore_model_level_stdv(num_states);

    for(int st=0; st<num_states; st++){
        auto params = event_sequences[0].pore_model->states[0]; //let's just initially get the params for AAAAAA
        pore_model_level_log_stdv[st] = params.level_log_stdv;
        pore_model_level_mean[st] = params.level_mean;
        pore_model_level_stdv[st] = params.level_stdv;
    }


    float* poreModelLevelLogStdvDev;
    hipMalloc( (void**)&poreModelLevelLogStdvDev, pore_model_level_log_stdv.size() * sizeof(float));
    hipMemcpyAsync( poreModelLevelLogStdvDev, pore_model_level_log_stdv.data(), pore_model_level_log_stdv.size() * sizeof(float), hipMemcpyHostToDevice );

    float* poreModelLevelMeanDev;
    hipMalloc( (void**)&poreModelLevelMeanDev, pore_model_level_mean.size() * sizeof(float));
    hipMemcpyAsync( poreModelLevelMeanDev, pore_model_level_mean.data(), pore_model_level_mean.size() * sizeof(float), hipMemcpyHostToDevice );

    float* poreModelLevelStdvDev;
    hipMalloc( (void**)&poreModelLevelStdvDev, pore_model_level_stdv.size() * sizeof(float));
    hipMemcpyAsync( poreModelLevelStdvDev, pore_model_level_stdv.data(), pore_model_level_stdv.size() * sizeof(float), hipMemcpyHostToDevice );


    float* eventsPerBaseDev;
    hipMalloc( (void**)&eventsPerBaseDev, eventsPerBase.size() * sizeof(float));
    hipMemcpyAsync( eventsPerBaseDev, eventsPerBase.data(), eventsPerBase.size() * sizeof(float), hipMemcpyHostToDevice );

    float* eventMeansDev;
    hipMalloc( (void**)&eventMeansDev, eventMeansSize);
    hipMemcpyAsync( eventMeansDev, eventMeans, eventMeansSize, hipMemcpyHostToDevice ); //malloc is taking 300us

    int* numRowsDev;
    hipMalloc( (void**)&numRowsDev, n_rows.size() * sizeof(int));
    hipMemcpyAsync( numRowsDev, n_rows.data(), n_rows.size() * sizeof(int), hipMemcpyHostToDevice );

    int* kmerRanksDev;
    int* kmerRanksRCDev;
    hipMalloc( (void**)&kmerRanksDev, kmer_ranks.size() * sizeof(int));
    hipMalloc( (void**)&kmerRanksRCDev, kmer_ranks_rc.size() * sizeof(int));
    hipMemcpyAsync( kmerRanksDev, kmer_ranks.data(), kmer_ranks.size() * sizeof(int), hipMemcpyHostToDevice );
    hipMemcpyAsync( kmerRanksRCDev, kmer_ranks_rc.data(), kmer_ranks_rc.size() * sizeof(int), hipMemcpyHostToDevice );

    int* eventStartsDev;
    hipMalloc( (void**)&eventStartsDev, e_starts.size() * sizeof(int));
    hipMemcpyAsync( eventStartsDev, e_starts.data(), e_starts.size() * sizeof(int), hipMemcpyHostToDevice );

    int* eventStridesDev;
    hipMalloc( (void**)&eventStridesDev, event_strides.size() * sizeof(int));
    hipMemcpyAsync( eventStridesDev, event_strides.data(), event_strides.size() * sizeof(int), hipMemcpyHostToDevice );

    int* eventOffsetsDev;
    hipMalloc( (void**)&eventOffsetsDev, eventOffsets.size() * sizeof(int));
    hipMemcpyAsync( eventOffsetsDev, eventOffsets.data(), eventOffsets.size() * sizeof(int), hipMemcpyHostToDevice );

    int num_blocks = n_states / PSR9_NUM_STATES;
    uint32_t num_kmers = num_blocks - 2; // two terminal blocks

    dim3 dimBlock(num_blocks - 2);
    dim3 dimGrid(1); // One thread per state, not including Start and Terminal state.

    float * returnValues;
    hipMalloc((void **) &returnValues, sizeof(float) * num_reads); //one score per read

    float* returnedValues;// = new float[num_reads];
    //size_t eventMeansSize = numEventsTotal * sizeof(float);
    hipHostAlloc(&returnedValues, num_reads * sizeof(float) , hipHostMallocDefault);

    getScores<<<dimGrid, dimBlock>>>(eventMeansDev,
            eventsPerBaseDev,
            numRowsDev,
            eventStartsDev,
            eventStridesDev,
            kmerRanksDev,
            kmerRanksRCDev,
            eventOffsetsDev,
            poreModelLevelLogStdvDev,
            poreModelLevelStdvDev,
            poreModelLevelMeanDev,
            returnValues);

    //hipDeviceSynchronize();
    hipMemcpyAsync(returnedValues, returnValues, num_reads *sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(eventMeansDev);
    hipFree(eventsPerBaseDev);
    hipFree(numRowsDev);
    hipFree(eventStartsDev);
    hipFree(eventStridesDev);
    hipFree(kmerRanksDev);
    hipFree(kmerRanksRCDev);
    hipFree(eventOffsetsDev);
    hipFree(poreModelLevelLogStdvDev);
    hipFree(poreModelLevelStdvDev);
    hipFree(poreModelLevelMeanDev);

    //Free host memory
    hipHostFree(eventMeans);

    float r = 0.0;
    for(int i=0; i<num_reads;i++){
        r += returnedValues[i];
    }

    return r;
}

std::vector<double> GpuAligner::variantScoresThresholded(std::vector<Variant> input_variants,
                                                        Haplotype base_haplotype,
                                                        std::vector<HMMInputData> event_sequences,
                                                        uint32_t alignment_flags,
                                                        int screen_score_threshold,
                                                        std::vector<std::string> methylation_types) {
    int numVariants = input_variants.size();

    std::vector<Variant> out_variants = input_variants;
    std::vector<Haplotype> variant_haplotypes(numVariants, base_haplotype);

    //loop over the vector, applying the variants to the haplotypes
    for (int i = 0; i<input_variants.size();i++){
        variant_haplotypes[i].apply_variant(input_variants[i]);
    }

    // Make methylated versions of each input sequence. Once for the base haplotype and once each for each variant
    std::vector<HMMInputSequence> base_sequences = generate_methylated_alternatives(base_haplotype.get_sequence(),
                                                                                    methylation_types);
    std::vector<std::vector<HMMInputSequence>> variant_sequences;
    for (auto v: variant_haplotypes){
        auto variant_sequence = generate_methylated_alternatives(v.get_sequence(), methylation_types);
        variant_sequences.push_back(variant_sequence);
    }

    assert(base_sequences.size() == 1);

    // return the sum of the score for the base sequences over all the event sequences
    double base_score = scoreKernel(base_sequences, event_sequences, alignment_flags);

    std::vector<double> v(variant_sequences.size());
    for (int i=0; i<variant_sequences.size(); i++){
        double score = scoreKernel(variant_sequences[i], event_sequences, alignment_flags); //TODO: Base sequence needs to be replaced with the variant itself
        v[i] = (score - base_score);
    }

    return v;
}
